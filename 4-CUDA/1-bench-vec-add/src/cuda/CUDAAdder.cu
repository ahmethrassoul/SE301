#include "hip/hip_runtime.h"
#include "CUDAAdder.hpp"
#include <iostream>

CUDAAdder::CUDAAdder(const int elements)
{
    nElements = elements;

    hipError_t err = hipSuccess;

    host_A = (float *)malloc( nElements * sizeof(float) );
    host_B = (float *)malloc( nElements * sizeof(float) );
    host_C = (float *)malloc( nElements * sizeof(float) );

    err = hipMalloc((void **)&gpu_A, nElements * sizeof(float));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&gpu_B, nElements * sizeof(float));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&gpu_C, nElements * sizeof(float));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
 
}
/*
 *
 *
 *##############################################################################
 *
 *
 */
extern __global__ void vectorAdd(const float *A, const float *B, float *C, int numElements);
/*
 *
 *
 *##############################################################################
 *
 *
 */
void CUDAAdder::execute()
{
    //
    // Launch the Vector Add CUDA Kernel
    //
    int threadsPerBlock = 256;
    int blocksPerGrid = (nElements + threadsPerBlock - 1) / threadsPerBlock;
    //printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(gpu_A, gpu_B, gpu_C, nElements);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //
    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    //
    err = hipMemcpy(host_C, gpu_C, nElements * sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}
/*
 *
 *
 *##############################################################################
 *
 *
 */
float* CUDAAdder::ptr_C()
{
    return host_C;
}
/*
 *
 *
 *##############################################################################
 *
 *
 */

int CUDAAdder::N()
{
    return nElements;
}
/*
 *
 *
 *##############################################################################
 *
 *
 */
void CUDAAdder::load_A(const float* src)
{
    int    length = N();
    for(int x = 0; x < length; x += 1)
        host_A[x] = src[x];

    hipError_t err = hipMemcpy(gpu_A, host_A, nElements * sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}
/*
 *
 *
 *##############################################################################
 *
 *
 */
void CUDAAdder::load_B(const float* src)
{
    int    length = N();
    for(int x = 0; x < length; x += 1)
        host_B[x] = src[x];

    hipError_t err = hipMemcpy(gpu_B, host_B, nElements * sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}
/*
 *
 *
 *##############################################################################
 *
 *
 */
void CUDAAdder::store_C(float* dst)
{
    float* ptr    = ptr_C();
    int    length = N();
    for(int x = 0; x < length; x += 1)
        dst[x] = ptr[x];
}
/*
 *
 *
 *##############################################################################
 *
 *
 */