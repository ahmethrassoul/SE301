#include "hip/hip_runtime.h"
/*
CPP translation of original Objective-C CUDAMax.m. Some stuff has been moved over to
the header. Source: https://developer.apple.com/documentation/metal/performing_calculations_on_a_gpu?language=objc

Original distribution license: LICENSE-original.txt.

Abstract:
A class to manage all of the Metal objects this app creates.
*/

#include "CUDAMax.hpp"
#include <iostream>
#include <hip/hip_runtime.h>
#include "helper/hip/hip_runtime_api.h"

const int threadsPerBlock = 256;

CUDAMax::CUDAMax(const int elements)
{
    nElements = elements;
    
    nGroups   = (nElements + threadsPerBlock - 1) / threadsPerBlock;

}
/*
 *
 *
 *##############################################################################
 *
 *
 */
extern __global__ void reduce_max_float(const float *A, float *C, int numElements);
/*
 *
 *
 *##############################################################################
 *
 *
 */
void CUDAMax::execute()
{

}
/*
 *
 *
 *##############################################################################
 *
 *
 */
int CUDAMax::N()
{
    return nElements;
}
/*
 *
 *
 *##############################################################################
 *
 *
 */
void CUDAMax::load(const float* src)
{

}
/*
 *
 *
 *##############################################################################
 *
 *
 */
float CUDAMax::result()
{

}
/*
 *
 *
 *##############################################################################
 *
 *
 */
