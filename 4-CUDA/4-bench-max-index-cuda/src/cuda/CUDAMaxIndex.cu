#include "hip/hip_runtime.h"
/*
CPP translation of original Objective-C CUDAMaxIndex.m. Some stuff has been moved over to
the header. Source: https://developer.apple.com/documentation/metal/performing_calculations_on_a_gpu?language=objc

Original distribution license: LICENSE-original.txt.

Abstract:
A class to manage all of the Metal objects this app creates.
*/

#include "CUDAMaxIndex.hpp"
#include <iostream>

const int threadsPerBlock = 256;

CUDAMaxIndex::CUDAMaxIndex(const int elements)
{
    nElements = elements;
    
    nGroups   = (nElements + threadsPerBlock - 1) / threadsPerBlock;

}
/*
 *
 *
 *##############################################################################
 *
 *
 */
extern __global__ void reduce_max_index(
    const float*   array,
          float*   result_max,
          int32_t* result_idx,
    const int numElements);
;
/*
 *
 *
 *##############################################################################
 *
 *
 */
void CUDAMaxIndex::execute()
{

}
/*
 *
 *
 *##############################################################################
 *
 *
 */
int CUDAMaxIndex::N()
{
    return nElements;
}
/*
 *
 *
 *##############################################################################
 *
 *
 */
void CUDAMaxIndex::load(const float* src)
{

}
/*
 *
 *
 *##############################################################################
 *
 *
 */
int CUDAMaxIndex::result()
{

}
/*
 *
 *
 *##############################################################################
 *
 *
 */