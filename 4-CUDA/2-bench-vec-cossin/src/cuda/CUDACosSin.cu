#include "hip/hip_runtime.h"
#include "CUDACosSin.hpp"
#include <iostream>

CUDACosSin::CUDACosSin(const int elements)
{
    nElements = elements;

    hipError_t err = hipSuccess;

    host_A = (float *)malloc( nElements * sizeof(float) );
    host_B = (float *)malloc( nElements * sizeof(float) );
    host_C = (float *)malloc( nElements * sizeof(float) );

    err = hipMalloc((void **)&gpu_A, nElements * sizeof(float));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&gpu_B, nElements * sizeof(float));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&gpu_C, nElements * sizeof(float));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}
/*
 *
 *
 *##############################################################################
 *
 *
 */
extern __global__ void cossin_arrays(const float *A, const float *B, float *C, int numElements);
/*
 *
 *
 *##############################################################################
 *
 *
 */
void CUDACosSin::execute()
{
    //
    // Launch the Vector Add CUDA Kernel
    //
    int threadsPerBlock = 256;
    int blocksPerGrid = (nElements + threadsPerBlock - 1) / threadsPerBlock;

    cossin_arrays<<<blocksPerGrid, threadsPerBlock>>>(gpu_A, gpu_B, gpu_C, nElements);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    modified = true;
}
/*
 *
 *
 *##############################################################################
 *
 *
 */
float* CUDACosSin::ptr_C()
{
    return host_C;
}
/*
 *
 *
 *##############################################################################
 *
 *
 */

int CUDACosSin::N()
{
    return nElements;
}
/*
 *
 *
 *##############################################################################
 *
 *
 */
void CUDACosSin::load_A(const float* src)
{
    int    length = N();
    for(int x = 0; x < length; x += 1)
        host_A[x] = src[x];

    hipError_t err = hipMemcpy(gpu_A, host_A, nElements * sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}
/*
 *
 *
 *##############################################################################
 *
 *
 */
void CUDACosSin::load_B(const float* src)
{
    int    length = N();
    for(int x = 0; x < length; x += 1)
        host_B[x] = src[x];

    hipError_t err = hipMemcpy(gpu_B, host_B, nElements * sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}
/*
 *
 *
 *##############################################################################
 *
 *
 */
void CUDACosSin::store_C(float* dst)
{
    if( modified == true )
    {
        hipError_t err = hipMemcpy(host_C, gpu_C, nElements * sizeof(float), hipMemcpyDeviceToHost);
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }      
        modified = false;
    }

    float* ptr    = ptr_C();
    int    length = N();
    for(int x = 0; x < length; x += 1)
        dst[x] = ptr[x];
}
/*
 *
 *
 *##############################################################################
 *
 *
 */